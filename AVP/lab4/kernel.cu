#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <cstdlib>
#include <windows.h>
#include <time.h>
#define MATRIX_SIZE 10



__host__ void printMatrix(int* m)
{
	for (int i = 0; i < MATRIX_SIZE; i++) {
		for (int j = 0; j < MATRIX_SIZE; j++) {
			printf("%4d", m[i * MATRIX_SIZE + j]);
		}
		printf("\n");
	}
}

__global__ void crossAdd(int* m, int* r, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (!(i == 0 || i == size - 1 || j == 0 || j == size - 1))
		*(r + i * MATRIX_SIZE + j) = *(m + i * MATRIX_SIZE + j) + *(m + i * MATRIX_SIZE + j + 1) + *(m + i * MATRIX_SIZE + j - 1) + *(m + (i + 1) * MATRIX_SIZE + j) + *(m + (i - 1) * MATRIX_SIZE + j);
}

__host__ bool equal(int* m1, int* m2)
{
	bool flag = true;
	for (int i = 0; (i < MATRIX_SIZE * MATRIX_SIZE) && flag; i++) {
		flag = (*(m1 + i) == *(m2 + i));
	}
	return flag;
}


__host__  void CPU(int * m, int * r) {
	for (int i = 1; i < MATRIX_SIZE - 1; i++) {
		for (int j = 1; j < MATRIX_SIZE - 1; j++) {
			*(r + i * MATRIX_SIZE + j) = *(m + i * MATRIX_SIZE + j) + *(m + i * MATRIX_SIZE + j + 1) + *(m + i * MATRIX_SIZE + j - 1) + *(m + (i + 1) * MATRIX_SIZE + j) + *(m + (i - 1) * MATRIX_SIZE + j);
		}
	}
}

__host__ void main() {
	srand(time(NULL));

	int* matrix = (int*)malloc(sizeof(int) * MATRIX_SIZE * MATRIX_SIZE);
	int* resCPU = (int*)calloc(sizeof(int), MATRIX_SIZE * MATRIX_SIZE);
	int* resCUDA = (int*)calloc(sizeof(int), MATRIX_SIZE * MATRIX_SIZE);


	for (int i = 0; i < MATRIX_SIZE; i++) {
		for (int j = 0; j < MATRIX_SIZE; j++) {
			matrix[i * MATRIX_SIZE + j] = rand() % 100;
		}
	}

	printMatrix(matrix);

	int start_time = GetTickCount();
	CPU(matrix, resCPU);
	printf("CPU time: %i\n", GetTickCount() - start_time);

	printMatrix(resCPU);


//Cuda mem
	int* c_matrix, *c_res;
	hipMalloc(&c_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	hipMalloc(&c_res, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	
	hipMemcpy(c_matrix, metrix,  MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpy(c_res, resCUDA,  MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
//CUDA 
	dim3 threadsPerBlock(MATRIX_SIZE);
	dim3 numBlocks(MATRIX_SIZE * MATRIX_SIZE * sizeof(int) / threadsPerBlock.x);
	
	hipEvent_t start, stop;
	float c_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	crossAdd<<<numBlocks, threadsPerBlock>>>(c_matrix, c_res);
	hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);
	hipEventElapsedTime(&c_time, start, stop);
	print("CUDA time: %.0f\n", c_time);
	
	hipMemcpy(resCUDA, c_res,  MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(c_matrix);
	cadaFree(c_res);

	printMAtrix(resCUDA);
}
